#include "hip/hip_runtime.h"
#include <stdlib.h> 
#include <assert.h> 
#include <float.h> 
#include <math.h>
#include <stdio.h>
#include <time.h>

#define THREAD_DIM 256

#define CHECK(call) { \
		const hipError_t error = call; \
		if (error != hipSuccess) { \
		printf("Error: %s:%d, ", __FILE__, __LINE__); \
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(1); \
	} \
}

void plot(double *data_points, int n, int m, int *labels, int k);
void countClusters(int *count, int k, int *labels, int n);
void init_centroids(double *data, int d, int k, double *centroids);

// O(n)+O(k) = O(n)
void countClusters(int *count, int k, int *labels, int n){
	int i;
	for(i = 0; i < k; i++){
		count[i]=0;
	}

	for(int j = 0; j < n; j++){
		count[labels[j]]+=1;
	}
};

__device__ double euclidean_distance(int d, double *point1, double *point2){
	double distance = 0;
	int j;
	for(j = 0; j < d; j++){
		distance += sqrt(powf(point1[j] - point2[j], 2));
	}
	return distance;
}

__device__ double atomicAddD(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// Complessità: O(kd)
__global__ void finding_closest(double *data, int n, int d, double *centroids, int k, int *labels, double *min_distances, double *tmp_centroids, int *counts ){
	
	int dim = d;
	int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0;
	double min_distance = DBL_MAX;
	double newDistance = 0;
	double *idata;
	int best_cluster = 0;
	double *c;

	if( thread_index < n ){

		idata = &data[thread_index * dim];

		for(j = 0; j < k; j++){
			//calcolo distanza tra data e clusters
			newDistance = euclidean_distance(dim, idata, &centroids[j * dim]);
		
			if(newDistance < min_distance){
				min_distance = newDistance;
				best_cluster = j;
			}
		}
		min_distances[thread_index] = min_distance;
		labels[thread_index] = best_cluster;

		c = &tmp_centroids[best_cluster * dim];
		for(int i=0;i<dim;i++){
			atomicAddD(&c[i], idata[i]);
		}
  		atomicAdd(&counts[best_cluster], 1);
	}
}

void init_centroids(double *data, int d, int k, double *centroids){
	double *ci;
	double *di;
	int i,j,h;
	for(i = 0, h= 5; i < k; i++, h += 5*i){
		ci = &centroids[i * d];
		di = &data[i * d];
		for (j = 0; j < d; j++){
			ci[j] = di[j];
		}
	}
}

int main(int argc, char *argv[]) {

	#define NUM_POINTS  5000
	#define THRESHOLD 1e-40
	#define DATASET_NAME "data/dataset.txt"
	#define DIM 2

	int i = 0, j = 0, k;
	double a = 0, b = 0;
	
    FILE *file;
    
    double *host_tmp_centroids;
    double *dev_tmp_centroids;
    double *host_min_distances;
  	
	double *host_centroids;
	double *host_data_points;
    int *host_labels;
    int *host_counts;

	double *dev_min_distances;
	double *dev_data_points;
	double *dev_centroids;
	int *dev_labels;
    int *dev_counts;

	// Contiene il numero di punti appartenenti al i-esimo cluster
	//int *count;


	if( argc == 2 ) {
      printf("Numero di cluster %s\n", argv[1]);
      printf("Numero di punti   %d\n", NUM_POINTS);
   	}else{
   		return -1;
   	}

   	k = atoi(argv[1]);

   	if( k > NUM_POINTS){
   		printf("ERRORE: il numero di cluster è superiore al numero di punti\n");
   		return -1;
   	}

   	//Allocazione della memoria HOST
	//count = (int*)calloc( k, sizeof(int) );
	host_labels = (int*)calloc(NUM_POINTS, sizeof(int));
	host_data_points = (double*)malloc(NUM_POINTS*DIM*sizeof(double));
	host_centroids = (double*)malloc(k*DIM*sizeof(double));
	host_min_distances = (double*)calloc(NUM_POINTS, sizeof(double));
	host_tmp_centroids = (double*)malloc(k*DIM*sizeof(double));
	host_counts = (int*)malloc(k*sizeof(int));

	// Allocazione della memoria DEVICE
	hipMalloc( (void**)&dev_min_distances, NUM_POINTS*sizeof(double) );
	hipMalloc( (void**)&dev_data_points, NUM_POINTS*DIM*sizeof(double) );
	hipMalloc( (void**)&dev_centroids, k*DIM*sizeof(double) );
	hipMalloc( (int**)&dev_labels, NUM_POINTS*sizeof(int) );
	hipMalloc( (int**)&dev_tmp_centroids, k*DIM*sizeof(double) );
	hipMalloc( (int**)&dev_counts, k*sizeof(int) );
	
	// Apre il dataset salvato nel file specificato e lo carica in memoria
	file = fopen(DATASET_NAME,"r");
	i=0;
  	while (fscanf(file, "%lf %lf", &a, &b) != EOF && i < NUM_POINTS*DIM) {
  		host_data_points[i] = a;
  		host_data_points[i+1] = b;
  		i += DIM;
	}

	// Inizializzazione dei centroidi
	init_centroids(host_data_points, DIM, k, host_centroids);

	// Copia dei dati dalla memoria HOST alla memoria DEVICE
	CHECK(hipMemcpy(dev_data_points, host_data_points, NUM_POINTS*DIM*sizeof(double), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_labels, host_labels, NUM_POINTS * sizeof(int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_min_distances, host_min_distances, NUM_POINTS * sizeof(double), hipMemcpyHostToDevice));
	

	//Calcola la distanza tra i dati ed i cluster
	double old_error;
	double error = DBL_MAX;
	int cycle_counter = 0;

	clock_t begin = clock();
	do {
		
		cycle_counter++;
		old_error = error;
		error = 0;

		CHECK(hipMemcpy(dev_centroids, host_centroids, k*DIM*sizeof(double), hipMemcpyHostToDevice));
		CHECK(hipMemset(dev_tmp_centroids, 0, k*DIM*sizeof(double)));
		CHECK(hipMemset(dev_counts, 0, k*sizeof(int)));

		//O(kd)
		finding_closest<<<20,256>>>(dev_data_points, NUM_POINTS, DIM, dev_centroids, k, dev_labels, dev_min_distances, dev_tmp_centroids, dev_counts);
	
		CHECK(hipDeviceSynchronize());
		
		CHECK(hipMemcpy(host_min_distances, dev_min_distances, NUM_POINTS * sizeof(double), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(host_labels, dev_labels, NUM_POINTS * sizeof(int), hipMemcpyDeviceToHost));
 		CHECK(hipMemcpy(host_tmp_centroids, dev_tmp_centroids, k * DIM * sizeof(double), hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(host_counts, dev_counts, k * sizeof(int), hipMemcpyDeviceToHost));

		CHECK(hipDeviceSynchronize());

		for(i=0;i<NUM_POINTS;i++){
			error += host_min_distances[i];
		}

		//O(n)
		
		// Calcolo dei nuovi centroidi
		double *tc;
		double *centroid;
		for(i=0; i<k; i++){
			centroid = &host_centroids[i * DIM];
			tc = &host_tmp_centroids[i * DIM];
			for(j=0; j<DIM; j++){
				if(host_counts[i]>0){
					centroid[j] = tc[j] / host_counts[i];
				}
			}
		}

		//O(kd)
		
	} while(fabs(error-old_error) > THRESHOLD);

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLK_TCK;
	printf("%lf\n",time_spent );
	
	// Disegna il grafico con gnuplot
	plot(host_data_points, NUM_POINTS, DIM, host_labels, k);

	


	// liberazione memoria host
	free(host_labels);
	free(host_data_points);
	free(host_centroids);
	free(host_min_distances);
	free(host_tmp_centroids);
	free(host_counts);
	
	// Liberazione memoria device
	hipFree(dev_min_distances);
	hipFree(dev_data_points);
	hipFree(dev_centroids);
	hipFree(dev_labels);
	hipFree(dev_tmp_centroids);
	hipFree(dev_counts);

	return 0;
}


void plot(double *data_points, int n, int m, int *labels, int k){
	int i = 0;
	int j = 0;

	#define NUM_COMMANDS 2
	char * commandsForGnuplot[] = {"set title \"Parallel k-means\"", "plot 'data.temp' u 1:2:3:3 with labels tc palette"};
	
	FILE * temp = fopen("data/plot/data.temp", "w");

	FILE * gnuplotPipe = _popen ("gnuplot -persistent", "w");
	
	for(i=0; i < n;i++){
		double *tmp = &data_points[i*m];
		for(j=0;j<m;j++){
			fprintf(temp, "%lf ", tmp[j]); 
		}
		fprintf(temp, "%d\n",labels[i]);
	}

	for (i=0; i < NUM_COMMANDS; i++){
		fprintf(gnuplotPipe, "%s \n", commandsForGnuplot[i]);
	}
}